#include "hip/hip_runtime.h"
//GPUTestMaxClique.cu

#include <stdio.h>
#include "cliqueMerging.cuh"

int main(int argc, char** argv)
{
    hipError_t retval = hipSuccess;
    struct clique_node* cliques = new clique_node();
    retval = findMaxCliquesGPU("test max clique on gpu", argc, argv, &cliques);

    //printKCliques(cliques);
    delete cliques;
    hipDeviceReset();

    return 0;
}
